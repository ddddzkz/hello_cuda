#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel()
{
	printf("hello cuda\n");
	__syncthreads();
}

int main()
{
	kernel<<<1, 1>>>();
	hipDeviceSynchronize();
	int i = 0, a = 0;
	for(i = 0; i < 10000000; i ++)
		a ++;
	printf("a = %d\n", a);
	return 0;
}
